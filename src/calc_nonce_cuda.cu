#include "hip/hip_runtime.h"
#include "sha256.cuh"
#include "calc_nonce_cuda.cuh"

#include <hiprand/hiprand_kernel.h>

//debug main() off
#include<stdio.h>
#define DEBUG

/***string***/

__device__ int my_strlen(char *str){
	int i = 0;
	while (str[i++] != '\0');
	i--;
	return i;
}

__device__ char * my_strcpy(char *dest, const char *src){
	int i = 0;
	do {
	  dest[i] = src[i];}
	while (src[i++] != 0);
	return dest;
}

__device__ int my_strcmp(const char *str_a, const char *str_b){
	while(*str_a == *str_b){
        if(*str_a == '\0'){
            return 0;
        }
        ++str_a;
        ++str_b;
    }
    return 1;
}

/***string***/

__device__
char change_10to16_one(int x) {

	char c;

	if (x >= 0 && x <= 9) c = x + '0';
	else if (x >= 10 && x <= 15) c = x + 'a' - 10;

	return c;
}

__device__
void string_change(unsigned char *hash, char *string_hash, int cpylen){

    int i=0,j=0;

    do{
        j=2*i;
        string_hash[j] = change_10to16_one(((int)hash[i] / 16));
        if(j>cpylen-2) break;
        j++;
        string_hash[j] = change_10to16_one(((int)hash[i] % 16));
        i++;
    }while(j<cpylen-1);

    string_hash[j+1] = '\0';

}

__device__
void random_nonce(char *nonce,hiprandState &s){

	const char set[] = "0123456789abcdef";
	int randam;

	for(int i = 0; i < 8; i++){
		randam = hiprand(&s) % 16;
		nonce[i]= set[randam];
	}

}

__device__
void calc_SHA256(char *string, char *string_hash, int hashlen){

	BYTE buf[SHA256_BLOCK_SIZE];
	SHA256_CTX ctx;

	sha256_init(&ctx);
	sha256_update(&ctx, (const BYTE *)string, my_strlen(string));
	sha256_final(&ctx, buf);

	string_change(buf,string_hash,64);

	sha256_init(&ctx);
	sha256_update(&ctx, (const BYTE *)string_hash, 64);
	sha256_final(&ctx, buf);

	string_change(buf,string_hash,hashlen);

}

__global__
void calc_nonce_kernel(volatile bool *found, char *zero_size, char *block, char *nonce){

	BYTE blocknonce[157+8+1];
	char hash[65];

	//extern __shared__ char sub_zero_size[];
	//__shared__ char sub_block[157+1];

	char sub_nonce[9];
	my_strcpy(sub_nonce,nonce);

	int block_len=my_strlen(block);
	int zero_size_len=my_strlen(zero_size);

	hiprandState s;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(0, id, id, &s);

	my_strcpy((char *)blocknonce,block);

	do{
		
		random_nonce(sub_nonce,s);

		my_strcpy((char *)blocknonce+block_len,sub_nonce);

		calc_SHA256((char *)blocknonce,hash,zero_size_len);

		if(my_strcmp(hash,zero_size) == 0) {
			my_strcpy(nonce,sub_nonce);
			break;
		}

	} while(!(*found));

	*found = true;

	//for debug
	//printf("id=%d,blocknonce:%s\n",id,blocknonce);

}

void calc_nonce_host(const char *zero_size, const char *block, char *nonce){

	char *d_zero_size;
	char *d_block;
	char *d_nonce;

	bool *d_found;
	hipMalloc((void**)&d_found, sizeof(bool));
	hipMemset(d_found, false, sizeof(bool));

	hipMalloc((void**)&d_zero_size,sizeof(char) * strlen(zero_size)+1);
	hipMalloc((void**)&d_block, sizeof(char) * strlen(block)+1);
	hipMalloc((void**)&d_nonce, sizeof(char) * strlen(nonce)+1);

	hipMemcpy(d_zero_size, zero_size, sizeof(char) * strlen(zero_size)+1, hipMemcpyHostToDevice);
	hipMemcpy(d_block, block, sizeof(char) * strlen(block)+1, hipMemcpyHostToDevice);
	hipMemcpy(d_nonce, nonce, sizeof(char) * strlen(nonce)+1, hipMemcpyHostToDevice);

	//calc_nonce_kernel<<<1024,1,strlen(zero_size)+1>>>(d_found, d_zero_size, d_block, d_nonce);
	calc_nonce_kernel<<<1024,1>>>(d_found, d_zero_size, d_block, d_nonce);

	hipMemcpy(nonce, d_nonce, sizeof(char) * strlen(nonce)+1, hipMemcpyDeviceToHost);

	hipFree(d_zero_size);
	hipFree(d_block);
	hipFree(d_nonce);
	hipFree(d_found);

}


#ifndef DEBUG
int main(void){

	char zero[200]="0";
	char block[20]="aaa";
	char nonce[9]="00000000";

	calc_nonce_host(zero,block,nonce);

	printf("nonce:%s\n",nonce);

	return 0;

}
#endif